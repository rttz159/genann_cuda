#include "hip/hip_runtime.h"
#include <iostream>
#include <hip/hip_runtime.h>
#include <assert.h>
#include "genann_cuda.cuh"
#include "genann.h"

__global__ void tiled_mat_mul_kernel(float* A, float* B, float* C, int N1, int N2, int N3){
    assert(TILE_WIDTH == blockDim.x);
    assert(TILE_WIDTH == blockDim.y);
    
    int by = blockIdx.y;
    int bx = blockIdx.x; 

    int ty = threadIdx.y;
    int tx = threadIdx.x; 

    // Working on C[i,j]
    int i = TILE_WIDTH*by + ty;
    int j = TILE_WIDTH*bx + tx;

    // Allocating shared memory
    __shared__ float sh_A[TILE_WIDTH][TILE_WIDTH];
    __shared__ float sh_B[TILE_WIDTH][TILE_WIDTH];

    // Parallel mat mul
    float value = 0;
    for (int phase = 0; phase < ceil((float)N2/TILE_WIDTH); phase++){
        if ((i < N1) && ((phase*TILE_WIDTH+tx) < N2))
          sh_A[ty][tx] = A[(i)*N2 + phase*TILE_WIDTH+tx];
        else
          sh_A[ty][tx] = 0.0f;

        if (((phase*TILE_WIDTH + ty) < N2) && (j < N3))
          sh_B[ty][tx] = B[(phase*TILE_WIDTH + ty)*N3+j];
        else
          sh_B[ty][tx] = 0.0f;
        __syncthreads();

        // Dot product
        for (int k = 0; k < TILE_WIDTH; k++)
            value += sh_A[ty][k] * sh_B[k][tx];
        __syncthreads();
    }
    if ((i < N1) && (j < N3))
      C[i*N3+j] = value;
}

// Applying activation functions
__global__ void apply_activation_kernel(float* data, int size, int activation_type) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < size) {
        switch(activation_type){
            case 0:
                data[idx] = genann_act_linear_cuda(data[idx]);
                break;
            case 1:
                data[idx] = genann_act_threshold_cuda(data[idx]);
                break;
            case 2:
                data[idx] = genann_act_sigmoid_cuda(data[idx]);
                break;
        }
    }
}

// Set bias on the array
__global__ void set_bias(float* d_input, int bias_index) {
    d_input[bias_index] = -1.0;
}

void launch_activation_kernel(float* d_data, int size, int activation_function) {
    unsigned int block_size = 256;
    unsigned int grid_size = (size + block_size - 1) / block_size;
    apply_activation_kernel<<<grid_size, block_size>>>(d_data, size, activation_function);
    CUDA_CHECK(hipDeviceSynchronize());
}

float *genann_run_cuda(genann *ann, float *input){

    if (!ann || !input) {
        fprintf(stderr, "Error: ann or input is NULL in genann_run_cuda.\n");
        return NULL;
    }

    // Allocate device memory for input and output of each layer
    float* output; // memory layout: [inputs,outputs]
    float* weights;

    // Get information
    unsigned int num_total_neurons = ann->total_neurons;
    unsigned int num_total_weight = ann->total_weights;
    unsigned int num_input = ann->inputs;
    unsigned int num_hidden = ann->hidden;
    unsigned int num_output = ann->outputs;
    unsigned int num_hidden_layers = ann->hidden_layers;

    enum GenannActivationType activation_hidden_type = ann->activation_hidden_type;
    enum GenannActivationType activation_output_type = ann->activation_output_type;

    // Calculate the number of bias neuron that need to be pre-appended to the output array
    unsigned int num_neuron_append = num_hidden_layers + 1;

    // Allocate host memory for output
    float* host_output = (float*)malloc(num_output * sizeof(float));

    // Allocate initial input and output for each neuron on the device, bias also included
    CUDA_CHECK(hipMalloc((void**)&output, (num_total_neurons + num_neuron_append) * sizeof(float)));
    CUDA_CHECK(hipMemcpy(output, input, num_input * sizeof(float), hipMemcpyHostToDevice));
    
    // Set the bias neuron
    set_bias<<<1,1>>>(output, num_input);
    CUDA_CHECK(hipDeviceSynchronize());

    // Allocate the weights to the device
    CUDA_CHECK(hipMalloc((void**)&weights, num_total_weight * sizeof(float)));
    CUDA_CHECK(hipMemcpy(weights, ann->weight, num_total_weight * sizeof(float), hipMemcpyHostToDevice));

    // Device pointer arithemtic
    float* d_input = output;
    float* d_result = output + num_input + 1;
    float* d_weight = weights;

    // Dimension initialization for block dimension
    dim3 dim_block(TILE_WIDTH, TILE_WIDTH, 1);

    if(!num_hidden){
        dim3 dim_grid(ceil(num_output/(float)(TILE_WIDTH)), ceil((num_input + 1)/(float)(TILE_WIDTH)), 1); // Dimension initialization for grid dimension
        // Kernel execution with no hidden layer
        tiled_mat_mul_kernel<<<dim_grid, dim_block>>>(d_input, d_weight, d_result, 1, (num_input + 1), num_output);
        CUDA_CHECK(hipDeviceSynchronize());
        launch_activation_kernel(d_result, num_output, activation_output_type);
        CUDA_CHECK(hipDeviceSynchronize());
    }else{
        dim3 dim_grid(ceil(num_hidden/(float)(TILE_WIDTH)), ceil((num_input + 1)/(float)(TILE_WIDTH)), 1); // Dimension initialization for grid dimension
        
        // Kernel execution for input to hidden layer
        tiled_mat_mul_kernel<<<dim_grid, dim_block>>>(d_input, d_weight, d_result, 1, (num_input + 1), num_hidden);
        CUDA_CHECK(hipDeviceSynchronize());
        launch_activation_kernel(d_result, num_hidden, activation_hidden_type);
        CUDA_CHECK(hipDeviceSynchronize());
        d_input += (num_input + 1);
        d_weight += ((num_input + 1) * num_hidden);
        set_bias<<<1,1>>>(d_result, (num_hidden)); // Set the bias neuron
        CUDA_CHECK(hipDeviceSynchronize());
        d_result += (num_hidden + 1);


        // Kernel execution for subsequent hidden layer
        for(int i = 1; i < num_hidden_layers; i++){
            dim3 dim_grid_hidden(ceil(num_hidden/(float)(TILE_WIDTH)), ceil((num_hidden + 1)/(float)(TILE_WIDTH)), 1); // Dimension initialization for grid dimension
            
            tiled_mat_mul_kernel<<<dim_grid_hidden, dim_block>>>(d_input, d_weight, d_result, num_hidden, (num_hidden + 1), num_hidden);
            CUDA_CHECK(hipDeviceSynchronize());
            launch_activation_kernel(d_result, num_hidden, activation_hidden_type);
            CUDA_CHECK(hipDeviceSynchronize());
            d_input += (num_hidden + 1);
            d_weight += (num_hidden * num_hidden);
            set_bias<<<1,1>>>(d_result, (num_hidden)); // Set the bias neuron
            CUDA_CHECK(hipDeviceSynchronize());
            d_result += (num_hidden + 1);
        }

        dim3 dim_grid_output(ceil(num_output/(float)(TILE_WIDTH)), ceil((num_hidden + 1)/(float)(TILE_WIDTH)), 1); // Dimension initialization for grid dimension

        // Kernel execution for hidden to output layer
        tiled_mat_mul_kernel<<<dim_grid_output, dim_block>>>(d_input, d_weight, d_result, 1, (num_hidden + 1), num_output);
        CUDA_CHECK(hipDeviceSynchronize());
        launch_activation_kernel(d_result, num_output, activation_output_type);
        CUDA_CHECK(hipDeviceSynchronize());
    }

    CUDA_CHECK(hipMemcpy(host_output, d_result, num_output * sizeof(float), hipMemcpyDeviceToHost));

    CUDA_CHECK(hipFree(output));
    CUDA_CHECK(hipFree(weights));

    return host_output;
}